#include "hip/hip_runtime.h"
#include "boxBlur.cuh"
#include "image_io.hpp"

using namespace cv;
using namespace std;

__global__ void boxBlur_device(uchar3* d_input, uchar3* d_output, int width, int height, int kernelSize) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height) {
        float sumX = 0;
        float sumY = 0;
        float sumZ = 0;
        int count = 0;

        for (int i = -kernelSize; i <= kernelSize; i++) {
            for (int j = -kernelSize; j <= kernelSize; j++) {
                int curRow = row + i;
                int curCol = col + j;
                if (curRow >= 0 && curRow < height && curCol >= 0 && curCol < width) {
                    sumX += d_input[curRow * width + curCol].x;
                    sumY += d_input[curRow * width + curCol].y;
                    sumZ += d_input[curRow * width + curCol].z;
                    count++;
                }
            }
        }

        d_output[row * width + col].x = sumX / count;
        d_output[row * width + col].y = sumY / count;
        d_output[row * width + col].z = sumZ / count;

    }

}

void boxBlur_host(Mat& input_image, Mat& output_image, uchar3* d_input, uchar3* d_output, string path, int kernelSize, bool save) {

    // Copy the input image to the GPU
    hipMemcpy(d_input, input_image.ptr<uchar3>(), input_image.cols * input_image.rows * sizeof(uchar3), hipMemcpyHostToDevice);

    //Launch the CUDA kernel to apply the blur filter
    dim3 blockDim(32, 32);
    dim3 gridDim((input_image.cols + blockDim.x - 1) / blockDim.x, (input_image.rows + blockDim.y - 1) / blockDim.y);

    // Call the box filter kernel 
    boxBlur_device << <gridDim, blockDim >> > (d_input, d_output, input_image.cols, input_image.rows, kernelSize);

    // Copy the output image from the GPU
    hipMemcpy(output_image.ptr<uchar3>(), d_output, input_image.cols * input_image.rows * sizeof(uchar3), hipMemcpyDeviceToHost);


    // Save the output image to disk
    if (save) saveImage(path, output_image, "_" + to_string(kernelSize) + "_boxBlur");
}