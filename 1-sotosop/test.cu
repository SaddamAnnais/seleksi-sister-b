#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void sum_area_table(int* input, int* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int index = y * width + x;
        int sum = input[index];

        if (x > 0) {
            sum += output[index - 1];
        }

        if (y > 0) {
            sum += output[index - width];
        }

        if (x > 0 && y > 0) {
            sum -= output[index - width - 1];
        }

        output[index] = sum;
    }
}

void generate_input(int* input, int width, int height) {
    for (int i = 0; i < width * height; i++) {
        input[i] = rand() % 10;
    }
}

void print_output(int* output, int width, int height) {
    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            std::cout << output[y * width + x] << " ";
        }
        std::cout << std::endl;
    }
}

int main() {
    const int width = 5;
    const int height = 5;
    const int size = width * height * sizeof(int);

    int* input, * output;
    hipMallocManaged(&input, size);
    hipMallocManaged(&output, size);

    generate_input(input, width, height);

    dim3 threads_per_block(16, 16);
    dim3 num_blocks((width + threads_per_block.x - 1) / threads_per_block.x, (height + threads_per_block.y - 1) / threads_per_block.y);

    sum_area_table << <num_blocks, threads_per_block >> > (input, output, width, height);

    hipDeviceSynchronize();

    print_output(output, width, height);

    hipFree(input);
    hipFree(output);

    return 0;
}