#include "hip/hip_runtime.h"
#include "grayscale.cuh"
#include <hip/hip_runtime.h>
#include "image_io.hpp"

using namespace std;
using namespace cv;

__global__ void grayscale_device(uchar3* input, uchar3* output, int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height) {
        int idx = row * width + col;
        uchar3 pixel = input[idx];

        float output_pixel = float((pixel.x + pixel.y + pixel.z) / 3);
        output[idx].x = output_pixel;
        output[idx].y = output_pixel;
        output[idx].z = output_pixel;
    }
}

void grayscale_host(Mat& input_image, Mat& output_image, uchar3* d_input, uchar3* d_output, string path, bool save) {
    // Copy the input image to the GPU
    hipMemcpy(d_input, input_image.ptr<uchar3>(), input_image.cols * input_image.rows * sizeof(uchar3), hipMemcpyHostToDevice);

    // Define the CUDA kernel launch parameters
    dim3 blockDim(32, 32);
    dim3 gridDim((input_image.cols + blockDim.x - 1) / blockDim.x, (input_image.rows + blockDim.y - 1) / blockDim.y);

    // Invoke Device Kernel
    grayscale_device << <gridDim, blockDim >> > (d_input, d_output, input_image.cols, input_image.rows);

    // Copy the output image from the GPU
    hipMemcpy(output_image.ptr<uchar>(), d_output, input_image.cols * input_image.rows * sizeof(uchar3), hipMemcpyDeviceToHost);

    // Save the output image to disk
    if (save) saveImage(path, output_image, "_grayscale");
}