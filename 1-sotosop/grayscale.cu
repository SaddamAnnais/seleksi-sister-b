#include "hip/hip_runtime.h"
#include "grayscale.cuh"
#include <hip/hip_runtime.h>
#include "image_io.hpp"

using namespace std;
using namespace cv;

__global__ void grayscale_device(uchar3* input, uchar* output, int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (col < width && row < height) {
        int idx = row * width + col;
        uchar3 pixel = input[idx];
        output[idx] = float((pixel.x + pixel.y + pixel.z) / 3);
    }
}

void grayscale_host(Mat& image, string path) {
    // Allocate memory on the GPU
    uchar3* d_input;
    uchar* d_output;
    hipMalloc(&d_input, image.cols * image.rows * sizeof(uchar3));
    hipMalloc(&d_output, image.cols * image.rows * sizeof(uchar));

    // Copy the input image to the GPU
    hipMemcpy(d_input, image.ptr<uchar3>(), image.cols * image.rows * sizeof(uchar3), hipMemcpyHostToDevice);

    // Define the CUDA kernel launch parameters
    dim3 blockDim(32, 32);
    dim3 gridDim((image.cols + blockDim.x - 1) / blockDim.x, (image.rows + blockDim.y - 1) / blockDim.y);

    // Invoke Device Kernel
    grayscale_device << <gridDim, blockDim >> > (d_input, d_output, image.cols, image.rows);

    // Copy the output image from the GPU
    Mat output_buffer(image.rows, image.cols, CV_8UC1);
    hipMemcpy(output_buffer.ptr<uchar>(), d_output, image.cols * image.rows * sizeof(uchar), hipMemcpyDeviceToHost);

    // Save the output image to disk
    saveImage(path, output_buffer, "_grayscale");
    output_buffer.copyTo(image);

    // Free memory on the GPU
    hipFree(d_input);
    hipFree(d_output);
}