#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "image_io.hpp"

using namespace std;
using namespace cv;


__global__ void contrast_device(uchar3* input, uchar3* output, int width, int height, int c) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height) {
        int idx = row * width + col;
        uchar3 pixel = input[idx];

        int r = max(0, min(255, c * (pixel.x - 128) + 128));
        int g = max(0, min(255, c * (pixel.y - 128) + 128));
        int b = max(0, min(255, c * (pixel.z - 128) + 128));

        output[idx].x = r;
        output[idx].y = g;
        output[idx].z = b;
    }
}

void contrast_host(Mat& image, string path, int c) {
    // Allocate memory on the GPU
    uchar3* d_input;
    uchar3* d_output;
    hipMalloc(&d_input, image.cols * image.rows * sizeof(uchar3));
    hipMalloc(&d_output, image.cols * image.rows * sizeof(uchar3));

    // Copy the input image to the GPU
    hipMemcpy(d_input, image.ptr<uchar3>(), image.cols * image.rows * sizeof(uchar3), hipMemcpyHostToDevice);

    // Define the CUDA kernel launch parameters
    dim3 blockDim(32, 32);
    dim3 gridDim((image.cols + blockDim.x - 1) / blockDim.x, (image.rows + blockDim.y - 1) / blockDim.y);

    // Launch the CUDA kernel to apply the contrast filter
    contrast_device << <gridDim, blockDim >> > (d_input, d_output, image.cols, image.rows, c);

    // Copy the output image from the GPU
    Mat output_buffer(image.rows, image.cols, CV_8UC3);
    hipMemcpy(output_buffer.ptr<uchar3>(), d_output, image.cols * image.rows * sizeof(uchar3), hipMemcpyDeviceToHost);

    // Save the output image to disk
    saveImage(path, output_buffer, "_" + to_string(c) + "_contrast");
    output_buffer.copyTo(image);

    // Free memory on the GPU
    hipFree(d_input);
    hipFree(d_output);
}
