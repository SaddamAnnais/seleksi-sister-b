#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "image_io.hpp"

using namespace std;
using namespace cv;


__global__ void contrast_device(uchar3* input, uchar3* output, int width, int height, int c) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height) {
        int idx = row * width + col;
        uchar3 pixel = input[idx];

        int r = max(0, min(255, c * (pixel.x - 128) + 128));
        int g = max(0, min(255, c * (pixel.y - 128) + 128));
        int b = max(0, min(255, c * (pixel.z - 128) + 128));

        output[idx].x = r;
        output[idx].y = g;
        output[idx].z = b;
    }
}

void contrast_host(Mat& input_image, Mat& output_image, uchar3* d_input, uchar3* d_output, string path, int c, bool save) {

    // Copy the input image to the GPU
    hipMemcpy(d_input, input_image.ptr<uchar3>(), input_image.cols * input_image.rows * sizeof(uchar3), hipMemcpyHostToDevice);

    // Define the CUDA kernel launch parameters
    dim3 blockDim(32, 32);
    dim3 gridDim((input_image.cols + blockDim.x - 1) / blockDim.x, (input_image.rows + blockDim.y - 1) / blockDim.y);

    // Launch the CUDA kernel to apply the contrast filter
    contrast_device << <gridDim, blockDim >> > (d_input, d_output, input_image.cols, input_image.rows, c);

    // Copy the output image from the GPU
    hipMemcpy(output_image.ptr<uchar3>(), d_output, input_image.cols * input_image.rows * sizeof(uchar3), hipMemcpyDeviceToHost);

    // Save the output image to disk
    if (save) saveImage(path, output_image, "_" + to_string(c) + "_contrast");
}
