#include "hip/hip_runtime.h"
#include "sobel_detection.cuh"
#include "image_io.hpp"

using namespace cv;
using namespace std;

__constant__ int sobel_x[9] = { -1, 0, 1, -2, 0, 2, -1, 0, 1 };
__constant__ int sobel_y[9] = { -1, -2, -1, 0, 0, 0, 1, 2, 1 };

__global__ void sobelEdgeDetection(uchar3* input, uchar3* output, int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height) {
        int sum_col = 0;
        int sum_row = 0;

        for (int i = -1; i <= 1; i++) {
            for (int j = -1; j <= 1; j++) {
                int pixel_col = col + i;
                int pixel_row = row + j;
                int pixel_index = pixel_row * width + pixel_col;

                float pixel_gray = (input[pixel_index].x + input[pixel_index].y + input[pixel_index].z)/3;

                sum_col += sobel_x[(i + 1) * 3 + (j + 1)] * pixel_gray;
                sum_row += sobel_y[(i + 1) * 3 + (j + 1)] * pixel_gray;
            }
        }
        unsigned char final_pixel = static_cast<unsigned char>(sqrtf(static_cast<float>(sum_col * sum_col + sum_row * sum_row)));
        output[row * width + col] = make_uchar3(final_pixel, final_pixel, final_pixel);
    }
}

void sobelEdgeDetection_host(cv::Mat& input_image, cv::Mat& output_image, uchar3* d_input, uchar3* d_output, std::string path, bool save) {
    // Copy the input image to the GPU
    hipMemcpy(d_input, input_image.ptr<uchar3>(), input_image.cols * input_image.rows * sizeof(uchar3), hipMemcpyHostToDevice);

    //Launch the CUDA kernel to apply the blur filter
    dim3 blockDim(32, 32);
    dim3 gridDim((input_image.cols + blockDim.x - 1) / blockDim.x, (input_image.rows + blockDim.y - 1) / blockDim.y);

    // Call the box filter kernel 
    sobelEdgeDetection << <gridDim, blockDim >> > (d_input, d_output, input_image.cols, input_image.rows);

    // Copy the output image from the GPU
    hipMemcpy(output_image.ptr<uchar3>(), d_output, input_image.cols * input_image.rows * sizeof(uchar3), hipMemcpyDeviceToHost);


    // Save the output image to disk
    if (save) saveImage(path, output_image, "_sobelEdgeDetection");
}