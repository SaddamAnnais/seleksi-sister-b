#include "hip/hip_runtime.h"
#include "saturation.cuh"
#include <string>
#include <iostream>
#include <hip/hip_runtime.h>
#include "image_io.hpp"
#include <math.h>

using namespace std;
using namespace cv;



__device__ uchar3 hsl2rgb(HSLColor hsl) {
    // Convert the HSL color to RGB
    float c = (1 - abs(2 * hsl.l - 1)) * hsl.s;
    float x = c * (1 - abs(fmodf(hsl.h / 60, 2) - 1));
    float m = hsl.l - c / 2;

    float r, g, b;
    if (hsl.h >= 0 && hsl.h < 60) {
        r = c;
        g = x;
        b = 0;
    }
    else if (hsl.h >= 60 && hsl.h < 120) {
        r = x;
        g = c;
        b = 0;
    }
    else if (hsl.h >= 120 && hsl.h < 180) {
        r = 0;
        g = c;
        b = x;
    }
    else if (hsl.h >= 180 && hsl.h < 240) {
        r = 0;
        g = x;
        b = c;
    }
    else if (hsl.h >= 240 && hsl.h < 300) {
        r = x;
        g = 0;
        b = c;
    }
    else {
        r = c;
        g = 0;
        b = x;
    }

    r += m;
    g += m;
    b += m;

    return make_uchar3(static_cast<unsigned char>(r * 255), static_cast<unsigned char>(g * 255), static_cast<unsigned char>(b * 255));
}

__device__ HSLColor rgb2hsl(uchar3 rgb) {
    // Convert the RGB color to HSL
    float r = rgb.x / 255.0f;
    float g = rgb.y / 255.0f;
    float b = rgb.z / 255.0f;

    float cmax = max(max(r, g), b);
    float cmin = min(min(r, g), b);
    float delta = cmax - cmin;

    float h, s, l;
    if (delta == 0) {
        h = 0;
    }
    else if (cmax == r) {
        h = fmodf((g - b) / delta, 6);
    }
    else if (cmax == g) {
        h = (b - r) / delta + 2;
    }
    else {
        h = (r - g) / delta + 4;
    }

    h *= 60;

    l = (cmax + cmin) / 2;

    if (delta == 0) {
        s = 0;
    }
    else {
        s = delta / (1 - fabs(2 * l - 1));
    }

    return { h, s, l };
}

__global__ void saturation_kernel(uchar3* input, uchar3* output, int width, int height, int c) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height) {
        int idx = row * width + col;
        uchar3 pixel = input[idx];

        // Convert to hsl
        HSLColor pixel_hsl = rgb2hsl(pixel);

        // Multiply the saturation by c
        pixel_hsl.s *= c;

        // Convert the back to RGB
        uchar3 pixel_rgb = hsl2rgb(pixel_hsl);

        output[idx].x = pixel_rgb.x;
        output[idx].y = pixel_rgb.y;
        output[idx].z = pixel_rgb.z;
    }
}

void saturation_host(Mat& image, string path, int c) {
    // Allocate memory on the GPU
    uchar3* d_input;
    uchar3* d_output;
    hipMalloc(&d_input, image.cols * image.rows * sizeof(uchar3));
    hipMalloc(&d_output, image.cols * image.rows * sizeof(uchar3));

    // Copy the input image to the GPU memory
    hipMemcpy(d_input, image.ptr<uchar3>(0), image.cols * image.rows * sizeof(uchar3), hipMemcpyHostToDevice);

    // Set the block size and grid size
    dim3 blockSize(32, 32);
    dim3 gridSize((image.cols + blockSize.x - 1) / blockSize.x, (image.rows + blockSize.y - 1) / blockSize.y);

    // Launch the kernel
    saturation_kernel << <gridSize, blockSize >> > (d_input, d_output, image.cols, image.rows, c);

    // Copy the output image from the GPU
    Mat output_buffer(image.rows, image.cols, CV_8UC3);
    hipMemcpy(output_buffer.ptr<uchar3>(0), d_output, image.cols * image.rows * sizeof(uchar3), hipMemcpyDeviceToHost);

    // Save the output image to disk
    saveImage(path, output_buffer, "_" + to_string(c) + "_saturation");
    output_buffer.copyTo(image);

    // Free the GPU memory
    hipFree(d_input);
    hipFree(d_output);
}